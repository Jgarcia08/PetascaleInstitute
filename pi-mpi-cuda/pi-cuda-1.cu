/* Pi - CUDA version 1 - uses integers for CUDA kernels
 * Author: Aaron Weeden, Shodor, May 2015
 */

#include <hip/hip_runtime.h>
#include <stdio.h> /* fprintf() */
#include <float.h> /* DBL_EPSILON() */
#include <math.h> /* sqrt() */

__global__ void calculateAreas(const int numRects, const double width,
    double *dev_areas) {
  const int threadId = threadIdx.x;
  const double x = (threadId * width);
  const double heightSq = (1.0 - (x * x));
  const double height =
    /* Prevent nan value for sqrt() */
    (heightSq < DBL_EPSILON) ? (0.0) : (sqrt(heightSq));

  if (threadId < numRects) {
    dev_areas[threadId] = (width * height);
  }
}

void calculateArea(const int numRects, double *area) {
  double *areas = (double*)malloc(numRects * sizeof(double));
  double *dev_areas;
  int i = 0;
  hipError_t err;

  if (areas == NULL) {
    fprintf(stderr, "malloc failed!\n");
  }

  err = hipMalloc((void**)&dev_areas, (numRects * sizeof(double)));

  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
  }

  calculateAreas<<<1, numRects>>>(numRects, (1.0 / numRects), dev_areas);

  err = hipMemcpy(areas, dev_areas, (numRects * sizeof(double)),
    hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
  }

  (*area) = 0.0;
  for (i = 0; i < numRects; i++) {
    (*area) += areas[i];
  }

  hipFree(dev_areas);

  free(areas);
}
